#include "hip/hip_runtime.h"
#include "g2d.h"

__global__ void times_dt(int jtot,int ktot,int nvar,int nghost,double* s, double* dt){

  int j  = blockDim.x*blockIdx.x + threadIdx.x + nghost;
  int k  = blockDim.y*blockIdx.y + threadIdx.y + nghost;
  int v  = threadIdx.z;

  s   += j*nvar + k*jtot*nvar + blockIdx.z*jtot*ktot*nvar;
  dt  += j      + k*jtot      + blockIdx.z*jtot*ktot;

  if(j+nghost < jtot and k+nghost < ktot){
    // for(int v=0; v<nvar; v++){
    s[v] *= dt[0];
    // }
  }
}

void G2D::precondition(double* sin, double* sout){

  int nl     = nM*nRey*nAoa;
  int qcount = nl*jtot*ktot*nvar;
  int count4 = nl*jtot*ktot*4;

  if(sin != sout){
    HANDLE_ERROR( hipMemcpy(sout, sin, qcount*sizeof(double), hipMemcpyDeviceToDevice) );
  }

  this->zero_bc(sout);

  dim3 vthr(32,4,nvar);
  dim3 vblk;
  vblk.x = (jtot-1-nghost*2)/vthr.x+1;
  vblk.y = (ktot-1-nghost*2)/vthr.y+1;
  vblk.z = nl;

  times_dt<<<vblk,vthr>>>(jtot,ktot,nvar,nghost,sout,dt);
  
  // return;

  // Set viscosity based on stored Q
  if(this->eqns != EULER)     this->set_mulam(this->q[GPU]);
  if(this->eqns == TURBULENT) this->set_muturb(this->q[GPU]);

  // debug_print(87,3,0,q[GPU],5);

  // Mean-flow equations:
  this->dadi(sout);

  // Turb equation:
  if(eqns == TURBULENT){
    this->sa_adi(sout);
  }

  this->zero_bc(sout);

}
