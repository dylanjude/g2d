#include "hip/hip_runtime.h"
#include "g2d.h"
#include <cstdio>
#include <cstdlib>

#define BIG   1e100

__global__ void shift_q(double* q_old, double* q_new, int tot, int* lmap){
  int i    = blockDim.x * blockIdx.x + threadIdx.x;
  int lnew = blockIdx.z;
  int lold = lmap[lnew];
  if(i<tot) q_new[i+lnew*tot] = q_old[i+lold*tot];
}


void G2D::check_convergence(){

  int l;
  bool* done = new bool[nl];

  double drop;

  double fmin, fmax, fvary;

  double eps=1e-16;

  for(l=0; l<nl; l++){ 
    // assume we're not done
    done[l] = false;

    drop = log10(res0[l]/res[l]);

    fmin =  BIG;
    fmax = -BIG;
    for(int i=0; i<AVG_HIST; i++){
      fmin = std::min(fmin, fhist[l*AVG_HIST+i]);
      fmax = std::max(fmax, fhist[l*AVG_HIST+i]);
    }
    fvary = 200*(fmax-fmin)/(fmax+fmin+eps);

    // printf("# CASE : M=%9.3f, Alpha=%9.3f, Re=%16.8e, drop=%7.3f, f_vary=%8.3f\n",  
    // 	   machs[CPU][l], aoas[CPU][l], reys[CPU][l]*machs[CPU][l], drop, fvary);

    // First criteria: residual converges more than 6 orders
    if(drop > 6){
      done[l] = true;
      printf("# DONE : M=%9.3f, Alpha=%9.3f, Re=%16.8e, dropped 5 orders\n", 
    	     machs[CPU][l], aoas[CPU][l], reys[CPU][l]*machs[CPU][l]);
      continue;
    }

    // Second criteria: forces have not changed more than 0.1%
    if(fvary < 0.1 and drop > 4){
      done[l] = true;
      printf("# DONE : M=%9.3f, Alpha=%9.3f, Re=%16.8e, <0.1%% change in forces\n", 
    	     machs[CPU][l], aoas[CPU][l], reys[CPU][l]*machs[CPU][l]);
      continue;
    }

  }

  int* lmap = new int[nl];

  int ll=0;
  for(l=0; l<nl; l++){

    if(done[l]){
      // continue without incrementing ll ( and start shifting next time )
      continue;
    }

    lmap[ll] = l;

    if(l == ll){
      // we haven't hit any completed cases, increment ll and move on
      ll++;
      continue;
    }

    // we need to shift:
    machs[CPU][ll]   = machs[CPU][l];
    aoas[CPU][ll]    = aoas[CPU][l];
    reys[CPU][ll]    = reys[CPU][l];

    res_fname[ll]    = res_fname[l];
    forces_fname[ll] = forces_fname[l];
    cpcf_fname[ll]   = cpcf_fname[l];
    sol_fname[ll]    = sol_fname[l];

    res[ll]          = res[l];
    res0[ll]         = res0[l];

    ll++;

  }

  this->nl = ll;

  int* lmap_gpu = (int*)wrk;

  // copy the map to the GPU
  HANDLE_ERROR( hipMemcpy(lmap_gpu, lmap, nl*sizeof(int), hipMemcpyHostToDevice) );
  
  // copy the shifted M, Aoa, Rey values to the GPU (small data so it's ok)
  HANDLE_ERROR( hipMemcpy(this->machs[GPU], this->machs[CPU], nl*sizeof(double), hipMemcpyHostToDevice) );
  HANDLE_ERROR( hipMemcpy(this->aoas[GPU],  this->aoas[CPU],  nl*sizeof(double), hipMemcpyHostToDevice) );
  HANDLE_ERROR( hipMemcpy(this->reys[GPU],  this->reys[CPU],  nl*sizeof(double), hipMemcpyHostToDevice) );
  
  double* newq = this->s;
  
  dim3 thr(256,1,1);
  dim3 blk(1,1,nl);
  blk.x = (jtot*ktot*nvar-1)/thr.x+1;
  
  // Shift q into qtmp (which is s residual storage). 
  shift_q<<<blk,thr>>>(q[GPU],newq,jtot*ktot*nvar,lmap_gpu);

  // Then swap s and q pointers.
  this->s      = this->q[GPU];
  this->q[GPU] = newq;

  delete[] done;
  delete[] lmap;

}
