#include "hip/hip_runtime.h"
#include "g2d.h"

#define GROUP_MEANFLOW

__global__ void reorder_and_square(int jtot, int ktot, int nvar, int nghost, double* s, double* wrk){

  int j  = blockDim.x*blockIdx.x + threadIdx.x;
  int k  = blockDim.y*blockIdx.y + threadIdx.y;
  int v  = threadIdx.z;

  int lin_idx;

#ifdef GROUP_MEANFLOW
  lin_idx = (j + 
             k*(jtot-nghost*2) + 
	     v*(jtot-nghost*2)*(ktot-nghost*2) +
	     blockIdx.z*2*(jtot-nghost*2)*(ktot-nghost*2));
#else
  lin_idx = (j + 
	     k*(jtot-nghost*2) + 
	     v*(jtot-nghost*2)*(ktot-nghost*2) +
	     blockIdx.z*nvar*(jtot-nghost*2)*(ktot-nghost*2));
#endif

  j += nghost;
  k += nghost;

  if(j+nghost > jtot-1 or k+nghost > ktot-1) return;

  s += (j + k*jtot + blockIdx.z*jtot*ktot)*nvar;

#ifdef GROUP_MEANFLOW
  double sum=0;
  if(v==0){
    for(int vv=0; vv<nvar-1; vv++){
      sum += s[vv]*s[vv];
    }
  } else {
    sum = s[nvar-1]*s[nvar-1];
  }
  wrk[lin_idx] = sum;
#else
  wrk[lin_idx] = s[v]*s[v];  
#endif

}

__global__ void sum1(double* a, int n, double* b){
  
  extern __shared__ double ish1[];
  int tid = threadIdx.x;
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  int v = blockIdx.y;
  int l = blockIdx.z;

  // initially there are n values between each variable on each grid
  if(i<n){
    ish1[tid] = a[i + v*n + l*n*gridDim.y]; // gridDim.y is nvar
  } else {
    ish1[tid] = 0;
  }

  __syncthreads();

  // initial increment is half the block dimension,
  // then 1/4, 1/8 etc. ">>" is a shift operator for
  // fast binary operations. >>1 shifts by 1 bit is
  // equivalent to a divide by two.
  for(int s=blockDim.x/2; s>0; s>>=1){
    if(tid < s){
      ish1[tid] += ish1[tid+s];
    }
    __syncthreads();
  }

  // the next "n" will be the number of blocks we have in the x-direction, gridDim.x
  if(tid == 0){ 
    b[blockIdx.x + v*gridDim.x + l*gridDim.x*gridDim.y] = ish1[0];
  } 

}

void G2D::check_convergence(int istep, double* s){

  int nl        = nM*nRey*nAoa;
  // int qcount    = nl*jtot*ktot*nvar;
  int i, leftover, smem;
  int j, k, l, v;

#ifdef GROUP_MEANFLOW
  int nv = 2;
#else
  int nv = nvar;
#endif

  dim3 vthr(32,4,nv);
  dim3 vblk;
  vblk.x = (jtot-1-nghost*2)/vthr.x+1;
  vblk.y = (ktot-1-nghost*2)/vthr.y+1;
  vblk.z = nl;

  int pts = (jtot-nghost*2)*(ktot-nghost*2);

  int c=0;
  double* scratch1 = &wrk[c]; c+= pts*nv*nl;
  double* scratch2 = &wrk[c]; c+= pts*nv*nl;
 
  // --------------------------------------------------------------
  // double* scpu  = new double[jtot*ktot*nvar*nl];
  // HANDLE_ERROR( hipMemcpy(scpu, s, jtot*ktot*nvar*nl*sizeof(double), hipMemcpyDeviceToHost) );
  // double l2cpu, ss;
  // int ii;
  // int vcheck=1;
  // int dcnt=1;
  // for(l=0; l<nl; l++){
  //   printf("[cpu] %3d ", l);
  //   for(v=0; v<nvar; v++){
  //     l2cpu=0;
  //     for(k=nghost; k<ktot-nghost; k++){
  // 	for(j=nghost; j<jtot-nghost; j++){
  // 	  ss = scpu[(j + k*jtot + l*jtot*ktot)*nvar+v];
  // 	  // if(ii++<dcnt) printf("%16.8e ", ss*ss);
  // 	  l2cpu += ss*ss;
  // 	}
  //     }
  //     printf("%16.8e ",l2cpu);
  //   }
  //   printf("\n");
  // }
  // delete[] scpu;
  // --------------------------------------------------------------------

  reorder_and_square<<<vblk,vthr>>>(jtot,ktot,nvar,nghost,s,scratch1);

  // HANDLE_ERROR( hipMemcpy(scpu, scratch1, pts*nv*nl*sizeof(double), hipMemcpyDeviceToHost) );
  // for(l=0;l<nl;l++){
  //   printf("[cpu]_%3d ", l);
  //   for(v=0; v<nv; v++){
  //     l2cpu=0;
  //     for(i=0; i<pts; i++){
  // 	  ss     = scpu[i + v*pts + l*nv*pts];
  // 	  l2cpu += ss;
  //     }
  //     printf("%16.8e ",l2cpu);
  //   }
  //   printf("\n");
  // }

  dim3 threads(1,1,1), blocks(1,nv,nl);

  double* l2var = new double[nv*nl];

  int n     = pts;
  int power = min(9, (int)ceil(log2(n*1.0)));

  threads.x = pow(2,power);
  leftover = n;
  i = 0;
  while(leftover > 1){

    blocks.x = (leftover - 1)/ threads.x + 1;
    smem = threads.x*sizeof(double);

    if(i%2 == 0){
      sum1<<<blocks,threads,smem>>>(scratch1, leftover, scratch2);
    } else {
      sum1<<<blocks,threads,smem>>>(scratch2, leftover, scratch1);
    }
    i++;
    leftover = blocks.x;
  }
  
  if(i%2 == 1){
    HANDLE_ERROR(hipMemcpy(l2var,scratch2,nv*nl*sizeof(double),hipMemcpyDeviceToHost));
  } else {
    HANDLE_ERROR(hipMemcpy(l2var,scratch1,nv*nl*sizeof(double),hipMemcpyDeviceToHost));
  }

  // scratch1 += pts;

  // printf("[All Norm] %6d %16.8e\n", istep, sqrt(normsq/pts));

  // printf("%6d ",istep);

  for(l=0; l<nl; l++){
    printf("%6d %3d ", istep, l);
    for(v=0; v<nv; v++){
      printf("%16.8e ", sqrt(l2var[v + l*nv]));
    }
    // printf("# <-- caution squared! \n");
    printf(" # [gpu]\n");
  }

  // delete[] scpu;
  // delete[] scpu2;

  delete[] l2var;

}
