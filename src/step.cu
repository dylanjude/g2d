#include "hip/hip_runtime.h"
#include "g2d.h"

__global__ void set_dt(int jtot,int ktot,int nvar,int nghost,
		       double* q, double* dt, double* vol, double2* Sj, double2* Sk, double cfl){

  int j  = blockDim.x*blockIdx.x + threadIdx.x;
  int k  = blockDim.y*blockIdx.y + threadIdx.y;

  if(j>jtot-1 or k>ktot-1) return;

  q   += j*nvar + k*jtot*nvar + blockIdx.z*jtot*ktot*nvar;
  dt  += j      + k*jtot      + blockIdx.z*jtot*ktot;

  double irho = 1.0/q[0];
  double u    = q[1]*irho;
  double v    = q[2]*irho;
  double p    = (GAMMA - 1.0)*(q[3] - 0.5*q[0]*(u*u + v*v)); 
  double c2   = GAMMA*p*irho;

  int gidx = j + k*jtot;

  double uu   = u*Sj[gidx].x + v*Sj[gidx].y;
  double vv   = u*Sk[gidx].x + v*Sk[gidx].y;

  double xs2  = Sj[gidx].x*Sj[gidx].x + Sj[gidx].y*Sj[gidx].y;
  double ys2  = Sk[gidx].x*Sk[gidx].x + Sk[gidx].y*Sk[gidx].y;

  double xsc  = sqrt(c2*xs2);
  double ysc  = sqrt(c2*ys2);

  // double eigmax = abs(uu) + xsc + abs(vv) + ysc;
  double eigmax = abs(uu) + xsc + abs(vv) + ysc + sqrt(c2*vol[gidx]*vol[gidx]); // <-- last term to match 3d Garfield

  // int ib = (j+nghost < jtot and k+nghost < ktot);
  int ib = 1;//(j+nghost < jtot and k+nghost < ktot);

  dt[0] = ib*vol[gidx]*cfl/eigmax;

}

#define DBGJ 191
#define DBGK 50

__global__ void update_q(int jtot,int ktot,int nvar,int nghost, double* q, double* s){

  int j  = blockDim.x*blockIdx.x + threadIdx.x + nghost;
  int k  = blockDim.y*blockIdx.y + threadIdx.y + nghost;

  q  += j*nvar + k*jtot*nvar + blockIdx.z*jtot*ktot*nvar;
  s  += j*nvar + k*jtot*nvar + blockIdx.z*jtot*ktot*nvar;

  if(j+nghost < jtot and k+nghost < ktot){

    for(int v=0; v<nvar; v++){
      q[v] += s[v];
    }
  }

}

void G2D::go(){

  int nstep=2000;
  int resmod=10;
  if(nstep > 999){
    resmod = 50;
  } else if(nstep > 99){
    resmod = 10;
  } else {
    resmod = 1;
  }

  int nl     = nM*nRey*nAoa;
  int qcount = nl*jtot*ktot*nvar;

  dim3 thr(32,16,1);
  dim3 blk;
  blk.x = (jtot-1)/thr.x+1;
  blk.y = (ktot-1)/thr.y+1;
  blk.z = nl;

  double cfl = 20.0;

  for(istep=0; istep<nstep; istep++){

    HANDLE_ERROR( hipMemcpy(qp, q[GPU], qcount*sizeof(double), hipMemcpyDeviceToDevice) );

    set_dt<<<blk,thr>>>(jtot,ktot,nvar,nghost,q[GPU],dt,vol,Sj,Sk,cfl);
    
    this->compute_rhs(q[GPU],s);

    // debug_print(87,3,0,s,5);

    // if(istep==nstep-1){
    //   this->write_sols();
    // }

    if((istep+1) % resmod == 0){
      this->check_convergence(istep+1, s);
    }

    this->precondition(s,s);

    update_q<<<blk,thr>>>(jtot,ktot,nvar,nghost,q[GPU],s);

  }



}
