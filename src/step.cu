#include "hip/hip_runtime.h"
#include "g2d.h"

__global__ void set_dt(int jtot,int ktot,int nvar,int nghost,
		       double* q, double* dt, double* vol, double2* Sj, double2* Sk, double cfl){

  int j  = blockDim.x*blockIdx.x + threadIdx.x;
  int k  = blockDim.y*blockIdx.y + threadIdx.y;

  if(j>jtot-1 or k>ktot-1) return;

  q   += j*nvar + k*jtot*nvar + blockIdx.z*jtot*ktot*nvar;
  dt  += j      + k*jtot      + blockIdx.z*jtot*ktot;

  double irho = 1.0/q[0];
  double u    = q[1]*irho;
  double v    = q[2]*irho;
  double p    = (GAMMA - 1.0)*(q[3] - 0.5*q[0]*(u*u + v*v)); 
  double c2   = GAMMA*p*irho;

  int gidx = j + k*jtot;

  double uu   = u*Sj[gidx].x + v*Sj[gidx].y;
  double vv   = u*Sk[gidx].x + v*Sk[gidx].y;

  double xs2  = Sj[gidx].x*Sj[gidx].x + Sj[gidx].y*Sj[gidx].y;
  double ys2  = Sk[gidx].x*Sk[gidx].x + Sk[gidx].y*Sk[gidx].y;

  double xsc  = sqrt(c2*xs2);
  double ysc  = sqrt(c2*ys2);

  // double eigmax = abs(uu) + xsc + abs(vv) + ysc;
  double eigmax = abs(uu) + xsc + abs(vv) + ysc + sqrt(c2*vol[gidx]*vol[gidx]); // <-- last term to match 3d Garfield

  // int ib = (j+nghost < jtot and k+nghost < ktot);
  int ib = 1;//(j+nghost < jtot and k+nghost < ktot);

  dt[0] = ib*vol[gidx]*cfl/eigmax;

}

#define DBGJ 191
#define DBGK 50

__global__ void update_q(int jtot,int ktot,int nvar,int nghost, double* q, double* s){

  int j  = blockDim.x*blockIdx.x + threadIdx.x + nghost;
  int k  = blockDim.y*blockIdx.y + threadIdx.y + nghost;

  q  += j*nvar + k*jtot*nvar + blockIdx.z*jtot*ktot*nvar;
  s  += j*nvar + k*jtot*nvar + blockIdx.z*jtot*ktot*nvar;

  if(j+nghost < jtot and k+nghost < ktot){

    for(int v=0; v<nvar; v++){
      q[v] += s[v];
    }
  }

}

void G2D::go(){

  double cfl = 100.0;

  this->take_steps(100,1,cfl);
  this->take_steps(100,1,cfl);
  this->take_steps(100,1,cfl);
  this->take_steps(100,1,cfl);

}


void G2D::take_steps(int nstep, int nsub, double cfl0){

  int checkmod=10;

  int qcount = nl*jtot*ktot*nvar;

  dim3 thr(16,16,1);
  dim3 blk;
  blk.x = (jtot-1)/thr.x+1;
  blk.y = (ktot-1)/thr.y+1;
  blk.z = nl;

  double cfl  = cfl0;

  bool check;

  int nl0 = nM*nAoa*nRey;

  for(int i=0; i<nstep && istep<1000000 && nl>0; i++){

    HANDLE_ERROR( hipMemcpy(qp, q[GPU], qcount*sizeof(double), hipMemcpyDeviceToDevice) );

    // CFL Ramping (linear)
    if(istep < 100){
      cfl = 1.0 + (cfl0-1)*istep/100;
    } else {
      cfl = cfl0;
    }

    check = ((i+1) % checkmod == 0);

    set_dt<<<blk,thr>>>(jtot,ktot,nvar,nghost,q[GPU],dt,vol,Sj,Sk,cfl);

    for(int isub=0; isub<nsub; isub++){

      this->istep++; // increment global step count

      this->compute_rhs(q[GPU],s);

      if(check){
	this->compute_residual(s, isub);
      }

      // this->precondition(s,s);
      this->gmres(s);

      update_q<<<blk,thr>>>(jtot,ktot,nvar,nghost,q[GPU],s);

    }

    if(check){

      printf("Iteration %6d : running %4d conditions (%4d complete)\n", istep, nl, nl0-nl);

      // monitor forces
      this->check_forces();

      // close the residual files (after gmres runs, since gmres also prints to that file)
      for(int l=0; l<nl; l++){
	if(this->resfile[l]){
	  fclose(this->resfile[l]);
	  this->resfile[l]=NULL;
	}
      }

    }

    // done timestep loop
  }

  this->write_cpcf();
  this->write_sols();

  // monitor convergence
  this->check_convergence();


}
